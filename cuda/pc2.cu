// Copyright Supranational LLC

#include "poseidon.cu"
#include <util/debug_helpers.hpp>
#include "host_ptr_t.hpp"

#ifndef __CUDA_ARCH__

#include <filesystem>
#include <chrono>
#include "../src/planner.cpp"
#include "pc2.cuh"
#include "cuda_lambda_t.hpp"
#include <util/util.hpp>

pc2_t::pc2_t(column_reader_t& _reader, size_t _nodes_to_read, size_t _batch_size,
             size_t _stream_count, int _write_core) :
  reader(_reader),
  nodes_to_read(_nodes_to_read),
  batch_size(_batch_size),
  tree_c_address(NODE_COUNT / PARTITIONS, TREE_ARITY, NODE_SIZE, 0),
  tree_r_address(NODE_COUNT / PARTITIONS, TREE_ARITY, NODE_SIZE, TREE_R_LAYER_SKIPS + 1),
  stream_count(_stream_count),
  gpu_results_c(_batch_size * PARALLEL_SECTORS / TREE_ARITY * stream_count),
  gpu_results_r(_batch_size * PARALLEL_SECTORS / TREE_ARITY * stream_count),
  host_buf_storage(num_host_bufs * batch_size * PARALLEL_SECTORS),
  write_core(_write_core)
{
  assert (nodes_to_read % stream_count == 0);

  // Open all tree-c and tree-r files
  const char* tree_c_filename_template = "results/sc-02-data-tree-c-s-%03ld-%ld.dat";
  const char* tree_r_filename_template = "results/sc-02-data-tree-r-last-s-%03ld-%ld.dat";
  std::filesystem::create_directory("results");
  for (size_t i = 0; i < PARALLEL_SECTORS; i++) {
    for (size_t j = 0; j < PARTITIONS; j++) {
      const size_t MAX = 256;
      char fname[MAX];

      // tree-c
      snprintf(fname, MAX, tree_c_filename_template, i, j);
      tree_c_fds[i][j] = open(fname, O_RDWR | O_CREAT, (mode_t)0664);
      assert (tree_c_fds[i][j] != -1);
      lseek(tree_c_fds[i][j], tree_c_address.data_size() - 1, SEEK_SET);
      assert (write(tree_c_fds[i][j], "", 1) != -1);
      tree_c_files[i][j] = (uint8_t*)mmap(NULL, tree_c_address.data_size(),
                                          PROT_WRITE, MAP_SHARED, tree_c_fds[i][j], 0);
      if (tree_c_files[i][j] == MAP_FAILED) {
        perror("mmap failed for tree_c file");
        exit(1);
      }
      assert(madvise(tree_c_files[i][j], tree_c_address.data_size(), MADV_RANDOM) == 0);

      // tree-r
      snprintf(fname, MAX, tree_r_filename_template, i, j);
      tree_r_fds[i][j] = open(fname, O_RDWR | O_CREAT, (mode_t)0664);
      assert (tree_r_fds[i][j] != -1);
      lseek(tree_r_fds[i][j], tree_r_address.data_size() - 1, SEEK_SET);
      assert (write(tree_r_fds[i][j], "", 1) != -1);
      tree_r_files[i][j] = (uint8_t*)mmap(NULL, tree_r_address.data_size(),
                                          PROT_WRITE, MAP_SHARED, tree_r_fds[i][j], 0);
      if (tree_r_files[i][j] == MAP_FAILED) {
        perror("mmap failed for tree_r file");
        exit(1);
      }
      assert(madvise(tree_r_files[i][j], tree_r_address.data_size(), MADV_RANDOM) == 0);
    }
  }
  
  // Compute the final offset in the file for GPU data
  tree_address_t final_tree(stream_count, TREE_ARITY, sizeof(fr_t), 0);
  final_gpu_offset_c = tree_c_address.data_size() - final_tree.data_size();
  final_gpu_offset_r = tree_r_address.data_size() - final_tree.data_size();

  // Compute an offset table used for multiple partitions
  size_t nodes_per_stream = nodes_to_read / stream_count;
  size_t layer_offset = nodes_per_stream;
  while (layer_offset >= TREE_ARITY) {
    layer_offsets_c.push_back(layer_offset);
    layer_offset /= TREE_ARITY;
  }

  layer_offset = nodes_per_stream;
  for (size_t i = 0; i < TREE_R_LAYER_SKIPS + 1; i++) {
    layer_offset /= TREE_ARITY;
  }
  while (layer_offset >= TREE_ARITY) {
    layer_offsets_r.push_back(layer_offset);
    layer_offset /= TREE_ARITY;
  }

  // Create GPU poseidon hashers and streams
  size_t resource_id = 0;
  for (size_t i = 0; i < ngpus(); i++) {
    auto& gpu = select_gpu(i);
    poseidon_columns.push_back(new PoseidonCuda<COL_ARITY_DT>(gpu));
    poseidon_trees.push_back(new PoseidonCuda<TREE_ARITY_DT>(gpu));
      
    for (size_t j = 0; j < stream_count / ngpus(); j++) {
      resources.push_back(new gpu_resource_t(resource_id, gpu, nodes_per_stream, batch_size,
                                             reader.alloc_node_ios()));
      resource_id++;
    }
  }

  // Register the page buffer with the CUDA driver
  size_t page_buffer_size;
  page_buffer = reader.get_buffer(page_buffer_size);
  hipHostRegister(page_buffer, page_buffer_size, hipHostRegisterDefault);

  // Set up host side buffers for returning data
  host_bufs.resize(num_host_bufs);
  host_buf_pool.create(num_host_bufs);
  host_buf_to_disk.create(num_host_bufs);
  for (size_t i = 0; i < num_host_bufs; i++) {
    host_bufs[i].data = &host_buf_storage[i * batch_size * PARALLEL_SECTORS];
    host_buf_pool.enqueue(&host_bufs[i]);
  }
}

pc2_t::~pc2_t() {
  for (size_t i = 0; i < PARALLEL_SECTORS; i++) {
    for (size_t j = 0; j < PARTITIONS; j++) {
      munmap(tree_c_files[i][j], tree_c_address.data_size());
      close(tree_c_fds[i][j]);

      munmap(tree_r_files[i][j], tree_r_address.data_size());
      close(tree_r_fds[i][j]);
    }
  }
  while (resources.size() > 0) {
    gpu_resource_t* r = resources.back();
    select_gpu(r->gpu);
      
    delete r;
    resources.pop_back();
  }
  for (size_t i = 0; i < ngpus(); i++) {
    delete poseidon_columns[i];
    delete poseidon_trees[i];
  }
  hipHostUnregister(page_buffer);
}

void pc2_t::hash() {
  auto start = chrono::high_resolution_clock::now();
  for (size_t partition = 0; partition < PARTITIONS; partition++) {
    auto pstart_gpu = chrono::high_resolution_clock::now();
    hash_gpu(partition);
    auto pstop_gpu = chrono::high_resolution_clock::now();
    hash_cpu(partition, &(gpu_results_c[0]), tree_c_files, final_gpu_offset_c);
    hash_cpu(partition, &(gpu_results_r[0]), tree_r_files, final_gpu_offset_r);
    auto pstop_cpu = chrono::high_resolution_clock::now();
    uint64_t secs_gpu = std::chrono::duration_cast<
      std::chrono::seconds>(pstop_gpu - pstart_gpu).count();
    uint64_t secs_cpu = std::chrono::duration_cast<
      std::chrono::seconds>(pstop_cpu - pstop_gpu).count();
    printf("Partition %ld took %ld seconds (gpu %ld, cpu %ld)\n",
           partition, secs_gpu + secs_cpu, secs_gpu, secs_cpu);
  }
  auto stop = chrono::high_resolution_clock::now();
  uint64_t secs = std::chrono::duration_cast<
    std::chrono::seconds>(stop - start).count();

  size_t total_page_reads = nodes_to_read * PARTITIONS / NODES_PER_PAGE * LAYER_COUNT;
  printf("pc2 took %ld seconds utilizing %0.1lf iOPS\n",
         secs, (double)total_page_reads / (double)secs);
}

void pc2_t::hash_gpu(size_t partition) {
  assert (stream_count % ngpus() == 0);

  nodes_per_stream = nodes_to_read / stream_count;

  thread_pool_t pool(1);

  for (size_t i = 0; i < resources.size(); i++) {
    resources[i]->reset();
  }
  
  // Start a thread to process writes to disk
  atomic<bool> terminate = false;
  atomic<bool> disk_writer_done = false;
  pool.spawn([this, &terminate, &disk_writer_done]() {
    // TODO: enable config of core
    set_core_affinity(write_core);

    const size_t batch_size = 32;
    buf_to_disk_t* to_disk_batch[batch_size];
    size_t count = 0;
    
    while(!terminate || host_buf_to_disk.size() > 0) {
      buf_to_disk_t* to_disk = host_buf_to_disk.dequeue();
      if (to_disk != nullptr) {
        for (size_t i = 0; i < PARALLEL_SECTORS; i++) {
          memcpy(to_disk->dst[i], to_disk->src[i], to_disk->size);
        }
        host_buf_pool.enqueue(to_disk);
      }
    }
    disk_writer_done = true;
  });
  
  bool all_done = false;
  cuda_lambda_t cuda_notify(1);
  in_ptrs_d<TREE_ARITY> in_d;
  buf_to_disk_t* to_disk;
  buf_to_disk_t* to_disk_r;
  fr_t* fr;

  while (!all_done) {
    all_done = true;
    for (size_t resource_num = 0; resource_num < resources.size(); resource_num++) {
      gpu_resource_t& resource = *resources[resource_num];
      select_gpu(resource.gpu);
      int gpu_id = resource.gpu.id();
      fr_t* host_buf_c = (fr_t*)reader.get_buffer_id(resource.id);
      fr_t* host_buf_r = (fr_t*)reader.get_buffer_id(resources.size() + resource.id);

      if (resource.state != ResourceState::DONE) {
        all_done = false;
      }

      fr_t* out_c_d = nullptr;
      fr_t* out_r_d = nullptr;
      size_t layer_offset;
      node_id_t addr;
      size_t offset_c;
      size_t offset_r;
      uint64_t start_node;
      bool write_tree_r;

      // Device storage for the hash result
      if (resource.work_c.buf != nullptr) {
        out_c_d = &(*resource.work_c.buf)[0];
        out_r_d = &(*resource.work_r.buf)[0];
      }
      
      switch (resource.state) {
      case ResourceState::DONE:
        // Nothing
        break;
        
      case ResourceState::IDLE:
        // Initiate data read
        resource.last = !resource.scheduler_c.next([](work_item_t<gpu_buffer_t>& w) {},
                                                   &resource.work_c);
        resource.scheduler_r.next([](work_item_t<gpu_buffer_t>& w) {},
                                  &resource.work_r);
        if (resource.work_c.is_leaf) {
#ifdef DISABLE_FILE_READS
          resource.state = ResourceState::HASH_COLUMN;
          resource.column_data = reader.get_buffer_id(resource_num);
#else
          resource.state = ResourceState::DATA_READ;
#endif
        } else {
          resource.state = ResourceState::HASH_LEAF;
        }
        break;

      case ResourceState::DATA_READ:
        // Initiate the next data read
        start_node = ((uint64_t)resource.work_c.idx.node() * batch_size +
                      nodes_per_stream * resource.id +
                      partition * nodes_to_read);
        resource.column_data = reader.read_columns
          (start_node, resource.id, &resource.valid, &resource.valid_count, resource.node_ios);
        resource.state = ResourceState::DATA_WAIT;
        break;

      case ResourceState::DATA_WAIT:
        if (resource.valid.load() == resource.valid_count) {
          resource.state = ResourceState::HASH_COLUMN;
        }
        break;
      
      case ResourceState::HASH_COLUMN:
        to_disk = host_buf_pool.dequeue();
        if (to_disk == nullptr) {
          break;
        }
        
        resource.stream.HtoD(&resource.column_data_d[0], resource.column_data, resource.batch_elements);

        // Hash the columns
        poseidon_columns[gpu_id]->hash_batch_device
          (out_c_d, &resource.column_data_d[0], &resource.aux_d[0],
           batch_size * PARALLEL_SECTORS, PARALLEL_SECTORS,
           resource.stream, true, false, true, true);

        // Initiate copy of the hashed data from GPU
        fr = to_disk->data;
        resource.stream.DtoH(fr, out_c_d, batch_size * PARALLEL_SECTORS);

        // Initiate transfer of tree-c data to files
        layer_offset = layer_offsets_c[resource.work_c.idx.layer() - 1];
        addr = node_id_t(resource.work_c.idx.layer() - 1,
                         resource.work_c.idx.node() * batch_size + layer_offset * resource_num);
        offset_c = tree_c_address.address(addr);

        for (size_t i = 0; i < PARALLEL_SECTORS; i++) {
          to_disk->dst[i] = (fr_t*)&tree_c_files[i][partition][offset_c];
          to_disk->src[i] = &to_disk->data[i * batch_size];
        }
        to_disk->size = batch_size * sizeof(fr_t);

        resources[resource_num]->async_done = false;
        cuda_notify.schedule(resource.stream, [this, resource_num, to_disk, offset_c]() {
          this->host_buf_to_disk.enqueue(to_disk);
          resources[resource_num]->async_done = true;
        });

        resource.state = ResourceState::HASH_COLUMN_LEAVES;
        break;
        
      case ResourceState::HASH_COLUMN_LEAVES:
        if (!resources[resource_num]->async_done) {
          break;
        }
        to_disk = host_buf_pool.dequeue();
        if (to_disk == nullptr) {
          break;
        }

        // Hash tree-c
        poseidon_trees[gpu_id]->hash_batch_device
          (out_c_d, out_c_d, &resource.aux_d[0],
           batch_size * PARALLEL_SECTORS / TREE_ARITY, 1,
           resource.stream, false, false, true, true);

        // Hash tree-r using layer 11
        poseidon_trees[gpu_id]->hash_batch_device
          (out_r_d,
           &resource.column_data_d[batch_size * PARALLEL_SECTORS * (LAYER_COUNT - 1)],
           &resource.aux_d[0],
           batch_size * PARALLEL_SECTORS / TREE_ARITY,
           PARALLEL_SECTORS,
           resource.stream, false, true, true, true);

        // Initiate copy of the hashed data from GPU, reusing the host side column buffer
        resource.stream.DtoH(&to_disk->data[0], out_c_d,
                             batch_size * PARALLEL_SECTORS / TREE_ARITY);
        
        // Initiate transfer of tree-c data to files
        layer_offset = layer_offsets_c[resource.work_c.idx.layer()];
        addr = node_id_t(resource.work_c.idx.layer(),
                         resource.work_c.idx.node() * batch_size / TREE_ARITY +
                         layer_offset * resource_num);
        offset_c = tree_c_address.address(addr);
        for (size_t i = 0; i < PARALLEL_SECTORS; i++) {
          to_disk->dst[i] = (fr_t*)&tree_c_files[i][partition][offset_c];
          to_disk->src[i] = &to_disk->data[i * TREE_ARITY];
        }
        to_disk->size = TREE_ARITY * sizeof(fr_t);
        
        resources[resource_num]->async_done = false;
        cuda_notify.schedule(resource.stream, [this, resource_num, to_disk]() {
          this->host_buf_to_disk.enqueue(to_disk);
          resources[resource_num]->async_done = true;
        });

        resource.state = ResourceState::HASH_WAIT;
        break;

      case ResourceState::HASH_LEAF:
        if (host_buf_pool.size() < 2) {
          break;
        }
        to_disk = host_buf_pool.dequeue();
        assert (to_disk != nullptr);
        
        // Hash tree-c
        for (size_t i = 0; i < TREE_ARITY; i++) {
          in_d.ptrs[i] = &(*resource.work_c.inputs[i])[0];
        }
        poseidon_trees[gpu_id]->hash_batch_device_ptrs
          (out_c_d, in_d, &resource.aux_d[0],
           batch_size * PARALLEL_SECTORS / TREE_ARITY, 1,
           resource.stream, false, false, true, true);

        // Hash tree-r 
        for (size_t i = 0; i < TREE_ARITY; i++) {
          in_d.ptrs[i] = &(*resource.work_r.inputs[i])[0];
        }
        poseidon_trees[gpu_id]->hash_batch_device_ptrs
          (out_r_d, in_d, &resource.aux_d[0],
           batch_size * PARALLEL_SECTORS / TREE_ARITY, 1,
           resource.stream, false, false, true, true);

        // Initiate copy of the hashed data
        resource.stream.DtoH(&to_disk->data[0], out_c_d, batch_size * PARALLEL_SECTORS / TREE_ARITY);
        if (resource.last) {
          // Stash the final result in a known place
          fr_t* host_buf_c = (fr_t*)reader.get_buffer_id(resource_num);
          CUDA_OK(hipMemcpyAsync(host_buf_c, &to_disk->data[0],
                                  batch_size * PARALLEL_SECTORS / TREE_ARITY * sizeof(fr_t),
                                  hipMemcpyHostToHost, resource.stream));
        }

        // Compute offsets in the output files - tree-c
        layer_offset = layer_offsets_c[resource.work_c.idx.layer()];
        addr = node_id_t(resource.work_c.idx.layer(),
                         resource.work_c.idx.node() * batch_size / TREE_ARITY +
                         layer_offset * resource_num);
        offset_c = tree_c_address.address(addr);
        for (size_t i = 0; i < PARALLEL_SECTORS; i++) {
          to_disk->dst[i] = (fr_t*)&tree_c_files[i][partition][offset_c];
          to_disk->src[i] = &to_disk->data[i * TREE_ARITY];
        }
        to_disk->size = TREE_ARITY * sizeof(fr_t);

        // tree-r
        write_tree_r = resource.work_r.idx.layer() > TREE_R_LAYER_SKIPS;
        if (write_tree_r) {
          to_disk_r = host_buf_pool.dequeue();
          assert (to_disk_r != nullptr);
          resource.stream.DtoH(&to_disk_r->data[0], out_r_d,
                               batch_size * PARALLEL_SECTORS / TREE_ARITY);
          if (resource.last) {
            // Stash the final result in a known place
            fr_t* host_buf_r = (fr_t*)reader.get_buffer_id(resources.size() + resource_num);
            CUDA_OK(hipMemcpyAsync(host_buf_r, &to_disk_r->data[0],
                                    batch_size * PARALLEL_SECTORS / TREE_ARITY * sizeof(fr_t),
                                    hipMemcpyHostToHost, resource.stream));
          }

          layer_offset = layer_offsets_r[resource.work_r.idx.layer() - TREE_R_LAYER_SKIPS - 1];
          addr = node_id_t(resource.work_r.idx.layer() - TREE_R_LAYER_SKIPS - 1,
                           resource.work_r.idx.node() * batch_size / TREE_ARITY +
                           layer_offset * resource_num);
          offset_r = tree_r_address.address(addr);
          for (size_t i = 0; i < PARALLEL_SECTORS; i++) {
            to_disk_r->dst[i] = (fr_t*)&tree_r_files[i][partition][offset_r];
            to_disk_r->src[i] = &to_disk_r->data[i * TREE_ARITY];
          }
          to_disk_r->size = TREE_ARITY * sizeof(fr_t);
        }
        
        // Initiate transfer of data to files
        resources[resource_num]->async_done = false;
        cuda_notify.schedule(resource.stream, [this, resource_num,
                                               to_disk, to_disk_r, write_tree_r]() {
          this->host_buf_to_disk.enqueue(to_disk);
          if (write_tree_r) {
            this->host_buf_to_disk.enqueue(to_disk_r);
          }
          resources[resource_num]->async_done = true;
        });

        resource.state = ResourceState::HASH_WAIT;
        break;
      
      case ResourceState::HASH_WAIT:
        if (resource.async_done.load() == true) {
          if (resource.last) {
            resource.state = ResourceState::DONE;
          } else {
            resource.state = ResourceState::IDLE;
          }
        }
        break;

      default:
        abort();
      }
    }
  }
  for (size_t resource_num = 0; resource_num < stream_count; resource_num++) {
    resources[resource_num]->stream.sync();
  }

  terminate = true;

  size_t stride = batch_size * PARALLEL_SECTORS / TREE_ARITY;
  for (size_t resource_num = 0; resource_num < stream_count; resource_num++) {
    fr_t* host_buf_c = (fr_t*)reader.get_buffer_id(resource_num);
    memcpy(&gpu_results_c[resource_num * stride],
           &host_buf_c[0], batch_size * PARALLEL_SECTORS / TREE_ARITY * sizeof(fr_t));
  }
  for (size_t resource_num = 0; resource_num < stream_count; resource_num++) {
    fr_t* host_buf_r = (fr_t*)reader.get_buffer_id(resources.size() + resource_num);
    memcpy(&gpu_results_r[resource_num * stride],
           &host_buf_r[0], batch_size * PARALLEL_SECTORS / TREE_ARITY * sizeof(fr_t));
  }

  // Really only need this at the last partition...
  while (!disk_writer_done) {}
}

void pc2_t::hash_cpu(size_t partition, fr_t* input,
                     uint8_t* tree_files[PARALLEL_SECTORS][PARTITIONS],
                     size_t file_offset) {
  const size_t nodes_to_hash = stream_count; // Number of GPU streams
  
  tree_address_t final_tree(nodes_to_hash, TREE_ARITY, sizeof(fr_t), 0);

  Poseidon hasher(TREE_ARITY);

  auto hash_func = [this, &hasher, &final_tree, input, partition, tree_files, file_offset]
    (work_item_t<host_buffer_t>& w) {
    node_id_t addr(w.idx.layer() - 1, w.idx.node());
    size_t offset = final_tree.address(addr) + file_offset;

    const size_t stride = TREE_ARITY * sizeof(fr_t);
    if (w.is_leaf) {
      for (size_t sector = 0; sector < PARALLEL_SECTORS; sector++) {
        fr_t* out = &(*w.buf)[sector];
        fr_t in[TREE_ARITY];
        
        for (size_t i = 0; i < TREE_ARITY; i++) {
          in[i] = input[w.idx.node() * TREE_ARITY * PARALLEL_SECTORS +
                        sector * TREE_ARITY + i];
        }
        hasher.Hash((uint8_t*)out, (uint8_t*)in);
        memcpy(&tree_files[sector][partition][offset],
               &out[0], sizeof(fr_t));
      }
    } else {
      for (size_t sector = 0; sector < PARALLEL_SECTORS; sector++) {
        fr_t* out = &(*w.buf)[sector];
        fr_t in[TREE_ARITY];
        for (size_t i = 0; i < TREE_ARITY; i++) {
          in[i] = (*w.inputs[i])[sector];
        }
        hasher.Hash((uint8_t*)out, (uint8_t*)in);
        
        memcpy(&tree_files[sector][partition][offset],
               &out[0], sizeof(fr_t));
      }
    }
  };
  
  buffers_t<host_buffer_t> buffers(PARALLEL_SECTORS);
  scheduler_t<host_buffer_t> scheduler(nodes_to_hash, TREE_ARITY, buffers);
  scheduler.run(hash_func);
}

void pc2_hash(column_reader_t& reader, size_t nodes_to_read, size_t batch_size,
              size_t stream_count, int write_core) {
  pc2_t pc2(reader, nodes_to_read, batch_size, stream_count, write_core);
  pc2.hash();
}

#endif
